#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>

#define SIZE 200
#define BUCKETNUM 10

__global__ void insertionSort(int *b, int *index, int bucket_size, int n_buckets){
    
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_id <= n_buckets){

        int i, key, j;
        for (i = thread_id*bucket_size+1; i <= thread_id*bucket_size+index[thread_id]; i++) {
            key = b[i];
            j = i - 1;
    
            while (j >= thread_id*bucket_size && b[j] > key) {
                b[j + 1] = b[j];
                j = j - 1;
            }
            b[j + 1] = key;
        }

    }

}

void bucketSort(int arr[SIZE],int n){
    int max,min;
    minMax(arr,&min,&max);
    float value = (max - min) / (n-1);
    int range = ceil(value);

    int bucket_size = 3*(SIZE/n);
    int *b = (int*) malloc(n*bucket_size*sizeof(int));
    int *index = (int*) malloc(n*sizeof(int));
    for (int i=0;i<n;i++)
        index[i] = 0;
    
    double max_index = n-1;

    for (int i = 0; i < SIZE; i++) {
        int bi = (arr[i] - min) / range; // Index in bucket

        if (bi > max_index){
            bi = max_index;
        }
        b[bi*bucket_size+index[bi]] = arr[i];
        index[bi]++;
    }

    // sort the buckets cuda 
    int num_threads_per_block = 512;
    int num_blocks = ceil(n/num_threads_per_block);

    int *d_b;
    int *d_index;

    hipMalloc(&d_b, n*bucket_size*sizeof(int));
    hipMalloc(&d_index, n*sizeof(int));
    hipMemcpy(d_b,b,n*bucket_size*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_index, index, n*sizeof(int), hipMemcpyHostToDevice);

    insertionSort<<<num_blocks,num_threads_per_block>>>(d_b, d_index, bucket_size, n);

    hipDeviceSynchronize();

    free(b);
    free(index);
    hipFree(d_b);
    hipFree(d_index);

}

float minMax(int array[SIZE], int *min, int *max)
{
  int length;
  length = SIZE;
  int i = 1;
  *max = array[0];
  *min = array[0];

  for (i; i<length; i++) {
    if (*max < array[i])
      *max = array[i];

    if (*min > array[i])
      *min = array[i];
  }
 }
 
int main(){

    srand(time(NULL));
    int *arr = (int*) malloc(SIZE * sizeof(int));

    // fill array with random values 
    for(int i=0;i<SIZE;i++){
        arr[i] = rand()%100;
    }

    printf("Original array:\n");
    for (int i = 0; i < SIZE; i++)
        printf("%d ", arr[i]);
    printf("\n");

    bucketSort<<>>(arr, BUCKETNUM);

    printf("\nSorted array:\n");
    for (int i = 0; i < SIZE; i++)
        printf("%d ", arr[i]);
    printf("\n");

    return 0;
}
