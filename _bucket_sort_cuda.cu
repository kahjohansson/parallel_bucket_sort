#include "hip/hip_runtime.h"
#include <algorithm>
#include <iostream>
#include <vector>
#include <array>
#include <time.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
using namespace std;

#define SIZE 50
#define BUCKETNUM 50

// __global__ void SortBuckets(){

// }

int scatter(float arr[SIZE], int n){

    thrust::host_vector<thrust::host_vector<float>> buckets(n);
    float max = *max_element(arr, arr + SIZE);
    float min = *min_element(arr, arr + SIZE);
    float range = (max - min) / (n-1);

    // assign buckets for each element from arr (scatter step)
    for (int i = 0; i < n; i++) {
        int bi = (arr[i] - min) / range; // Index in bucket
        buckets[bi].push_back(arr[i]);
    }

    thrust::device_vector<thrust::device_vector<float>> device_buckets;

    return 0;
}

int main(){

    srand(time(NULL));

    // float *d_arr;

    float arr[SIZE];

    for(int i=0; i<SIZE; i++) {
        arr[i] = rand() % 1000;
    }

    scatter(arr, BUCKETNUM);

    // hipMalloc(&d_arr, SIZE*sizeof(float));

    // hipMemcpy(d_arr, arr, SIZE*sizeof(float), hipMemcpyHostToDevice);

    // BucketSort<<1,1>>();

    // hipMemcpy(arr, d_arr, SIZE*sizeof(float), hipMemcpyDeviceToHost);

    // hipFree(d_arr);

}