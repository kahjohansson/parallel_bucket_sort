
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>

#define SIZE 200
#define BUCKETNUM 10

__global__
void insertionSort(int *b, int *index, int bucket_size, int n_buckets)
{

    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_id <= n_buckets)
    {

        int i, key, j;
        for (i = thread_id * bucket_size; i < thread_id * bucket_size + index[thread_id]; i++)
        {
            key = b[i];
            j = i - 1;

            while (j >= thread_id * bucket_size && b[j] > key)
            {
                b[j + 1] = b[j];
                j = j - 1;
            }
            b[j + 1] = key;
        }
    }
}

__global__
void helloWorld(){
    printf("Hello world!");
}

void minMax(int array[SIZE], int *min, int *max)
{
    int length;
    length = SIZE;
    *max = array[0];
    *min = array[0];

    for (int i=1; i < length; i++)
    {
        if (*max < array[i])
            *max = array[i];

        if (*min > array[i])
            *min = array[i];
    }
}


int main()
{

    srand(time(NULL));
    int *arr = (int *)malloc(SIZE * sizeof(int));

    // fill array with random values
    for (int i = 0; i < SIZE; i++)
    {
        arr[i] = rand() % 100+1;
    }

    printf("Original array:\n");
    for (int i = 0; i < SIZE; i++)
        printf("%d ", arr[i]);
    printf("\n");

    // bucketSort(arr, BUCKETNUM);
    int n = BUCKETNUM;
    int max, min;
    minMax(arr, &min, &max);
    float value = (max - min) / (n - 1);
    int range = ceil(value);

    int bucket_size = 3 * (SIZE / n);
    int *b = (int *)malloc(n * bucket_size * sizeof(int));
    int *index = (int *)malloc(n * sizeof(int));
    for (int i = 0; i < n; i++)
        index[i] = 0;

    double max_index = n - 1;

    for (int i = 0; i < SIZE; i++)
    {
        int bi = (arr[i] - min) / range; // Index in bucket

        if (bi > max_index)
        {
            bi = max_index;
        }
        b[bi * bucket_size + index[bi]] = arr[i];
        index[bi]++;
    }

    for(int i=0; i < 3 * SIZE; i++){
        printf("%d ", b[i]);
    }
    // sort the buckets cuda
    int num_threads_per_block = 16;
    int num_blocks = ceil(n / num_threads_per_block);

    int *d_b;
    int *d_index;

    hipMalloc(&d_b, n * bucket_size * sizeof(int));
    hipMalloc(&d_index, n * sizeof(int));
    hipMemcpy(d_b, b, n * bucket_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_index, index, n * sizeof(int), hipMemcpyHostToDevice);

    insertionSort<<<4,4>>>(d_b, d_index, bucket_size, n);

    hipDeviceSynchronize();
 
    hipMemcpy(b, d_b, n * bucket_size * sizeof(int), hipMemcpyDeviceToHost);
 
 
    int k = 0;

    for(int i=0; i<n; i++){
        for(int j=0; j<index[i]; j++){
            arr[k] = b[bucket_size * i + j];
            k++; 
        }
    }
    free(b);
    free(index);
    hipFree(d_b);
    hipFree(d_index);

    printf("\nSorted array:\n");
    for (int i = 0; i < SIZE; i++)
        printf("%d ", arr[i]);
    printf("\n");
 
    free(arr);

    return 0;
}